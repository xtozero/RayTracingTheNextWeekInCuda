#include "hip/hip_runtime.h"
#include "AARect.h"
#include "Camera.h"
#include "Canvas.h"
#include "HittableList.h"
#include "Material.h"
#include "MovingSphere.h"
#include "Perlin.h"
#include "Random.h"
#include "Ray.h"
#include "Sphere.h"
#include "Vec3.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>

__device__ Color RayColor( hiprandState_t* randState, const Ray& r, HittableList** world, Color background, int depth )
{
	HitRecord rec;
	Color totalAttenuation( 1.0, 1.0, 1.0 );
	Ray curRay = r;

	for ( int i = 0; i < depth; ++i )
	{
		if ( ( *world )->Hit( curRay, 0.001, DBL_MAX, rec ) )
		{
			Color attenuation( 1.0, 1.0, 1.0 );
			Color emitted = rec.m_material->Emitted( rec.m_u, rec.m_v, rec.m_hitPosition );
			Ray scattered;

			if ( rec.m_material->Scatter( randState, curRay, rec, attenuation, scattered ) )
			{
				totalAttenuation = totalAttenuation * attenuation + emitted;
				curRay = scattered;
			}
			else
			{
				return totalAttenuation * emitted;
			}
		}
		else
		{
			return background;
		}
	}

	return Color( 0, 0, 0 );
}

__global__ void FillCanvas( Pixel* devPixels, HittableList** world, std::size_t width, std::size_t height, Color background, Camera cam, int numSample, int maxRayDepth )
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * width;
	y = height - y - 1; // invert y

	if ( x < width && y < height )
	{
		hiprandState_t randState;
		hiprand_init( offset, 0, 0, &randState );
		Color pixelColor( 0, 0, 0 );
		for ( int i = 0; i < numSample; ++i )
		{
			double u = double( x + RandomDouble( &randState ) ) / ( width - 1 );
			double v = double( y + RandomDouble( &randState ) ) / ( height - 1 );
			Ray r = cam.GetRay( &randState, u, v );
			pixelColor += RayColor( &randState, r, world, background, maxRayDepth );
		}
		
		WriteColor( devPixels[offset].m_color, pixelColor, numSample );
	}
}

__global__ void CreateWorld( HittableList** world )
{
	*world = new HittableList( );
	//( *world )->Add( new Sphere( Point3( 0, -100.5, -1 ), 100, new Lambertian( Color( 0.8, 0.8, 0.0 ) ) ) );
	//( *world )->Add( new Sphere( Point3( 0, 0, -1 ), 0.5, new Lambertian( Color( 0.1, 0.2, 0.5 ) ) ) );
	//( *world )->Add( new Sphere( Point3( -1, 0, -1 ), 0.5, new Dielectric( 1.5 ) ) );
	//( *world )->Add( new Sphere( Point3( -1, 0, -1 ), -0.45, new Dielectric( 1.5 ) ) );
	//( *world )->Add( new Sphere( Point3( 1, 0, -1 ), 0.5, new Metal( Color( 0.8, 0.6, 0.2 ), 0.0 ) ) );

	//double R = cos( HIP_PI / 4 );
	//( *world )->Add( new Sphere( Point3( -R, 0, -1 ), R, new Lambertian( Color( 0, 0, 1 ) ) ) );
	//( *world )->Add( new Sphere( Point3( R, 0, -1 ), R, new Lambertian( Color( 1, 0, 0 ) ) ) );

	( *world )->Add( new Sphere( Point3( 0, -1000, 0 ), 1000, new Lambertian( new CheckerTexture( Color( 0.2, 0.3, 0.1 ), Color( 0.9, 0.9, 0.9 ) ) ) ) );

	hiprandState_t randState;
	hiprand_init( 1024, 768, 0, &randState );
	for ( int i = -11; i < 11; ++i )
	{
		for ( int j = -11; j < 11; ++j )
		{
			double chooseMaterial = RandomDouble( &randState );
			Point3 center( i + 0.9 * RandomDouble( &randState ), 0.2, j + 0.9 * RandomDouble( &randState ) );

			if ( ( center - Point3( 4, 0.2, 0 ) ).Length( ) > 0.9 ) {
				if ( chooseMaterial < 0.8 ) {
					// diffuse
					Color albedo = Random( &randState ) * Random( &randState );
					Point3 center2 = center + Vec3( 0, RandomDouble( &randState, 0, 0.5 ), 0 );
					( *world )->Add( new MovingSphere( center, center2, 0.0, 1.0, 0.2, new Lambertian( albedo ) ) );
				}
				else if ( chooseMaterial < 0.95 ) {
					// metal
					Color albedo = Random( &randState, 0.5, 1 );
					double fuzz = RandomDouble( &randState, 0, 0.5 );
					( *world )->Add( new Sphere( center, 0.2, new Metal( albedo, fuzz ) ) );
				}
				else {
					// glass
					( *world )->Add( new Sphere( center, 0.2, new Dielectric( 1.5 ) ) );
				}
			}
		}
	}

	( *world )->Add( new Sphere( Point3( 0, 1, 0 ), 1.0, new Dielectric( 1.5 ) ) );

	( *world )->Add( new Sphere( Point3( -4, 1, 0 ), 1.0, new Lambertian( Color( 0.4, 0.2, 0.1 ) ) ) );

	( *world )->Add( new Sphere( Point3( 4, 1, 0 ), 1.0, new Metal( Color( 0.7, 0.6, 0.5 ), 0 ) ) );
}

__global__ void CreateTwoSpheresWorld( HittableList** world )
{
	*world = new HittableList( );

	( *world )->Add( new Sphere( Point3( 0, -10, 0 ), 10, new Lambertian( new CheckerTexture( Color( 0.2, 0.3, 0.1 ), Color( 0.9, 0.9, 0.9 ) ) ) ) );
	( *world )->Add( new Sphere( Point3( 0, 10, 0 ), 10, new Lambertian( new CheckerTexture( Color( 0.2, 0.3, 0.1 ), Color( 0.9, 0.9, 0.9 ) ) ) ) );
}

__global__ void CreatePerlinTextureWorld( HittableList** world, Perlin* perlin )
{
	*world = new HittableList( );

	( *world )->Add( new Sphere( Point3( 0, -1000, 0 ), 1000, new Lambertian( new NoiseTexture( perlin, 4 ) ) ) );
	( *world )->Add( new Sphere( Point3( 0, 2, 0 ), 2, new Lambertian( new NoiseTexture( perlin, 4 ) ) ) );
}

texture<uchar4, 2> g_earth;
__global__ void CreateEarthWorld( HittableList** world, int width, int height )
{
	*world = new HittableList( );

	( *world )->Add( new Sphere( Point3( 0, 0, 0 ), 2, new Lambertian( new ImageTexture( g_earth, width, height ) ) ) );
}

__global__ void CreateSimpleLightWorld( HittableList** world, Perlin* perlin )
{
	*world = new HittableList( );

	( *world )->Add( new Sphere( Point3( 0, -1000, 0 ), 1000, new Lambertian( new NoiseTexture( perlin, 4 ) ) ) );
	( *world )->Add( new Sphere( Point3( 0, 2, 0 ), 2, new Lambertian( new NoiseTexture( perlin, 4 ) ) ) );

	( *world )->Add( new XYRect( 3, 5, 1, 3, -2, new DiffuseLight( Color( 4, 4, 4 ) ) ) );
	( *world )->Add( new Sphere( Point3( 0, 7, 0 ), 2, new DiffuseLight( Color( 4, 4, 4 ) ) ) );
}

__global__ void DestroyWorld( HittableList** world )
{
	(*world)->Clear( );
	delete *world;
}

int main( )
{
	Point3 lookFrom( 13, 2, 3 );
	Point3 lookAt( 0, 0, 0 );
	double fov = 40.0;
	double aperture = 0.0;
	int SamplesPerPixel = 100;
	Color background( 0.7, 0.8, 1.0 );

	Perlin* perlinTexture = nullptr;

	HittableList** world = nullptr;
	hipMalloc( (void**)&world, sizeof( HittableList* ) );

	float3* deviceEarth = nullptr;

	switch ( 0 )
	{
	case 1:
		CreateWorld<<<1, 1>>>( world );
		fov = 20.0;
		aperture = 0.1;
		break;
	case 2:
		CreateTwoSpheresWorld<<<1, 1>>>( world );
		fov = 20.0;
		break;
	case 3:
		hipMalloc( &perlinTexture, sizeof( Perlin ) );
		GeneratePerlinTexture<<<16, 16>>>( perlinTexture );
		CreatePerlinTextureWorld<<<1, 1>>>( world, perlinTexture );
		fov = 20.0;
		break;
	case 4:
		{
			int width = 0;
			int height = 0;
			int componentPerPixel = 4;

			unsigned char* data = stbi_load( "earthmap.jpg", &width, &height, &componentPerPixel, componentPerPixel );

			hipMalloc( (void**)&deviceEarth, sizeof( uchar4 ) * width * height );
			hipMemcpy( deviceEarth, data, sizeof( uchar4 ) * width * height, hipMemcpyHostToDevice );

			delete[] data;

			hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>( );
			hipBindTexture2D( nullptr, &g_earth, deviceEarth, &desc, width, height, sizeof( uchar4 ) * width );

			CreateEarthWorld<<<1, 1>>>( world, width, height );
			fov = 20.0;
		}
		break;
	case 5:
	default:
		hipMalloc( &perlinTexture, sizeof( Perlin ) );
		GeneratePerlinTexture<<<16, 16>>>( perlinTexture );
		CreateSimpleLightWorld<<<1, 1>>>( world, perlinTexture );
		SamplesPerPixel = 400;
		background = Color( 0, 0, 0 );
		lookFrom = Point3( 26, 3, 6 );
		lookAt = Point3( 0, 2, 0 );
		fov = 20.0;
		break;
	}

	// camera
	constexpr double aspectRatio = 16.0 / 9.0;
	Camera cam( lookFrom, lookAt, fov, aspectRatio, aperture, 10, 0.0, 1.0 );

	// canvas
	constexpr int canvasWidth = 400;
	const int canvasHeight = static_cast<int>( canvasWidth / aspectRatio );
	Canvas canvas( canvasWidth, canvasHeight );

	Pixel* devPixels = nullptr;
	hipMalloc( (void**)&devPixels, canvas.Size( ) );

	int curDevice = 0;
	hipGetDevice( &curDevice );

	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop, curDevice );

	dim3 grids( static_cast<unsigned int>( ( canvas.Width() + 7 ) / 8 ) , static_cast<unsigned int>( ( canvas.Height( ) + 7 ) / 8 ) );
	dim3 threads( 8, 8 );
	FillCanvas<<<grids, threads>>>( devPixels, world, canvas.Width(), canvas.Height(), background, cam, SamplesPerPixel, 50 );

	DestroyWorld<<<1, 1>>>( world );

	hipMemcpy( canvas.Pixels(), devPixels, canvas.Size( ), hipMemcpyDeviceToHost );
	hipFree( devPixels );
	hipFree( world );
	hipFree( perlinTexture );
	hipFree( deviceEarth );

	hipUnbindTexture( &g_earth );

	canvas.WriteFile( "./image5_2.ppm" );
}